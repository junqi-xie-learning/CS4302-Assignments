/* File:
 *     rbf.cu
 *
 *
 * Idea:
 *     Computes a parallel rbf kernel of matrices with CUDA.  If not `tiled`,
 *     each thread will be responsible for calculating one element of the RBF
 *     kernel matrix.  If `tiled`, each thread will be responsible for loading
 *     part of the matrices into shared memory and then calculating part of the
 *     RBF kernel matrix.
 *     Note that the TILE_WIDTH should be tuned according to the workload and
 *     the GPU architecture to achieve the best performance.
 *
 * Compile:
 *     nvcc -o rbf.out rbf.cu
 * Usage:
 *     ./rbf.out <sigma> <dimension> <m> <n>
 *
 * Input:
 *     None unless compiled with debug mode.
 *     If in debug mode, read matrix `A`, `B` from standard input.
 * Output:
 *     Elapsed time for the computation
 *     If in debug mode, print the RBF kernel matrix.
 */

#include <iostream>
#include <random>
#include "hip/hip_runtime.h"
using namespace std;

bool debug = false;
bool tiled = true;
const int TILE_WIDTH = 16;

/*------------------------------------------------------------------
 * Function:  rbf_kernel
 * Purpose:   Kernel function to compute an element in the matrix produced by
 *            RBF kernels of row vectors of `A` and `B`.
 *            Note that `A_d`, `B_d`, and `C_d` are in the device memory.
 * In args:   A_d, B_d, sigma, dimension, m, n
 * Out arg:   C_d[row * n + col]
 */
template <typename T>
__global__ void rbf_kernel(T A_d[], T B_d[], T C_d[], T sigma, int dimension, int m, int n)
{
    // Calculate the row index of the working element in C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate the column index of the working element in C
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < m) && (col < n))
    {
        T dist{ };
        // Each thread computes one element of the block sub-matrix
        for (int i = 0; i < dimension; ++i)
        {
            T diff = A_d[row * dimension + i] - B_d[col * dimension + i];
            dist += diff * diff;
        }
        C_d[row * n + col] = exp(-dist / (2 * sigma * sigma));
    }
}

/*------------------------------------------------------------------
 * Function:  rbf
 * Purpose:   Wrapper function to compute the RBF kernel of `A` and `B`.
 *            where features are row vectors.
 * In args:   A, B, sigma, dimension, m, n
 * Out arg:   C
 */
template <typename T>
void rbf(T A[], T B[], T C[], T sigma, int dimension, int m, int n)
{
    int A_size = m * dimension * sizeof(T), B_size = n * dimension * sizeof(T),
        C_size = m * n * sizeof(T);
    T *A_d, *B_d, *C_d;

    // Transfer A and B to device memory
    hipMalloc((void **)&A_d, A_size);
    hipMemcpy(A_d, A, A_size, hipMemcpyHostToDevice);
    hipMalloc((void **)&B_d, B_size);
    hipMemcpy(B_d, B, B_size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    hipMalloc((void **)&C_d, C_size);

    // Kernel Invocation
    dim3 dimGrid(1, 1);
    dim3 dimBlock(m, n);
    rbf_kernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, sigma, dimension, m, n);

    // Transfer C from device to host
    hipMemcpy(C, C_d, C_size, hipMemcpyDeviceToHost);
    // Free device matrices
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

/*------------------------------------------------------------------
 * Function:  rbf_tiled_kernel
 * Purpose:   Kernel function to compute an element in the RBF kernel with
 *            optimization of tiling and utilization of shared memory.
 *            Note that `A_d`, `B_d`, and `C_d` are in the device memory.
 * In args:   A_d, B_d, sigma, dimension, m, n
 * Out arg:   C_d[row * k + col]
 */
template <typename T>
__global__ void rbf_tiled_kernel(T A_d[], T B_d[], T C_d[], T sigma, int dimension, int m, int n)
{
    __shared__ T A_shared[TILE_WIDTH][TILE_WIDTH];
    __shared__ T B_shared[TILE_WIDTH][TILE_WIDTH];

    // Identify the row and column of the working element in C
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * TILE_WIDTH + ty;
    int col = blockIdx.x * TILE_WIDTH + tx;

    // Loop over the A and B tiles required to compute P element
    T dist{ };
    for (int i = 0; i < dimension; i += TILE_WIDTH)
    {
        // Collaborative loading of M and N tiles into shared memory
        if ((row < m) && (i + tx) < dimension)
        {
            A_shared[ty][tx] = A_d[row * dimension + i + tx];
        }
        if ((i + ty) < dimension && (col < n))
        {
            B_shared[ty][tx] = B_d[(i + ty) * n + col];
        }
        __syncthreads();

        for (int ii = 0; ii < TILE_WIDTH; ++ii)
        {
            T diff = A_shared[ty][ii] - B_shared[ii][tx];
            dist += diff * diff;
        }
        __syncthreads();
    }

    if ((row < m) && (col < n))
    {
        C_d[row * n + col] = exp(-dist / (2 * sigma * sigma));
    }
}

/*------------------------------------------------------------------
 * Function:  rbf_tiled
 * Purpose:   Wrapper function of RBF kernel with optimization of tiling and
 *            utilization of shared memory.
 * In args:   A, B, sigma, dimension, m, n
 * Out arg:   C
 */
template <typename T>
void rbf_tiled(T A[], T B[], T C[], T sigma, int dimension, int m, int n)
{
    int A_size = m * dimension * sizeof(T), B_size = n * dimension * sizeof(T),
        C_size = m * n * sizeof(T);
    T *A_d, *B_d, *C_d;

    // Transfer A and B to device memory
    hipMalloc((void **)&A_d, A_size);
    hipMemcpy(A_d, A, A_size, hipMemcpyHostToDevice);
    hipMalloc((void **)&B_d, B_size);
    hipMemcpy(B_d, B, B_size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    hipMalloc((void **)&C_d, C_size);

    // Kernel Invocation
    dim3 dimGrid(TILE_WIDTH, TILE_WIDTH);
    dim3 dimBlock(ceil(m / (double)TILE_WIDTH), ceil(n / (double)TILE_WIDTH));
    rbf_tiled_kernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, sigma, dimension, m, n);

    // Transfer C from device to host
    hipMemcpy(C, C_d, C_size, hipMemcpyDeviceToHost);
    // Free device matrices
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

/*------------------------------------------------------------------
 * Function: generate_matrix
 * Purpose:  Use the random number generator random to generate
 *           the entries in `A` in [0, 0]
 * In arg:   m, n
 * Out arg:  A
 */
double *generate_matrix(int m, int n)
{
    default_random_engine generator;
    uniform_real_distribution<double> distribution{0, 0};

    double *A = new double[m * n];
    for (int i = 0; i < m * n; i++)
        A[i] = distribution(generator);
    return A;
}

/*------------------------------------------------------------------
 * Function: read_matrix
 * Purpose:  Read in a matrix
 * In arg:   m, n
 * Out arg:  A
 */
double *read_matrix(int m, int n)
{
    double *A = new double[m * n];
    for (int i = 0; i < m * n; i++)
        cin >> A[i];
    return A;
}

/*------------------------------------------------------------------
 * Function: print_matrix
 * Purpose:  Print a matrix
 * In args:  A, m, n
 */
void print_matrix(double *A, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
            cout << A[i * n + j] << " ";
        cout << endl;
    }
}

int main(int argc, char *argv[])
{
    // Get command line args
    double sigma = stod(argv[1]);
    int dimension = stoi(argv[2]), m = stoi(argv[3]), n = stoi(argv[4]);

    // Generate matrices
    double *A = nullptr, *B = nullptr;
    if (debug)
    {
        cout << "Enter matrix A: " << endl;
        A = read_matrix(dimension, m);
        cout << "Enter matrix B: " << endl;
        B = read_matrix(dimension, n);
    }
    else
    {
        cout << "Generated matrix A of size " << dimension << " * " << m << ", "
             << "matrix B of size " << dimension << " * " << n << endl;
        A = generate_matrix(dimension, m);
        B = generate_matrix(dimension, n);
    }

    // Initialize CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed;

    // Call `rbf` and get the time elapsed
    double *C = new double[m * n];
    hipEventRecord(start);

    if (tiled)
    {
        rbf_tiled(A, B, C, sigma, dimension, m, n);
    }
    else
    {
        rbf(A, B, C, sigma, dimension, m, n);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    cout << "RBF kernel calculated. Elapsed time: " << elapsed << " seconds" << endl;

    if (debug)
    {
        cout << "The RBF kernel is: " << endl;
        print_matrix(C, m, n);
    }

    delete[] A;
    delete[] B;
    delete[] C;
    return 0;
}
